/*
 * _ADD_V4_CU_
 *
 * 2022 Mert SIDE
 *
 * This file is a part of the CS5375 lectures at Texas Tech University.
 *
 * With unified memory
 *
 */


#include <hip/hip_runtime.h>
#include <iostream>
#include <math.h>

// CUDA kernel to initialize elements of two arrays
__global__ void init(int n, float *x, float *y) 
{
  int index = threadIdx.x + blockIdx.x * blockDim.x;
  int stride = blockDim.x * gridDim.x;
  for (int i = index; i < n; i += stride) {
    x[i] = 1.0f;
    y[i] = 2.0f;
  }
}

// CUDA kernel to add elements of two arrays
__global__
void add(int n, float *x, float *y) 
{
  int index = blockIdx.x * blockDim.x + threadIdx.x;
  int stride = blockDim.x * gridDim.x;
  for (int i = index; i < n; i += stride)
    y[i] = x[i] + y[i];
}

int main(void) 
{
  int N = 1<<25; // 33M elements
  //int N = 1<<20; // 1M elements
  float *x, *y;
 
  // Allocate Unified Memory -- accessible from CPU or GPU
  hipMallocManaged(&x, N*sizeof(float));
  hipMallocManaged(&y, N*sizeof(float));
 
  int blockSize = 256;
  int numBlocks = (N + blockSize - 1) / blockSize;  
  
  // initialize x and y arrays on the host
  init<<<numBlocks, blockSize>>>(N, x, y); 
  
  // Launch kernel on 33M elements on the GPU
  add<<<numBlocks, blockSize>>>(N, x, y);
 
  // Wait for GPU to finish before accessing on host
  hipDeviceSynchronize();
 
  // Check for errors (all values should be 3.0f)
  float maxError = 0.0f;
  for (int i = 0; i < N; i++)
    maxError = fmax(maxError, fabs(y[i]-3.0f));
  std::cout << "Max error: " << maxError << std::endl;
 
  // Free memory
  hipFree(x); 
  hipFree(y);
  
  return 0;
}
