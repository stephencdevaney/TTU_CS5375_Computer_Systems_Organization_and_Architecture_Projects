#include "hip/hip_runtime.h"
/*
 * _MATRIXMUL_GPU_CU_
 *
 * Orginal code by Mert SIDE
 * Updates by Stephen Devaney
 * 2022
 *
 * CS5375 Computer Systems Organization and Architecture 
 * Guest Lecture: GPU Programming
 *
 * Multiplying two matrices on the GPU
 *
 */


#include <iostream>
#include <stdio.h>
#include <stdlib.h>


// ------------------------------------------------------------------ init (unified memory) *********************************************** added by Stephen Devaney in part 4
__global__ void init(int N, double *x, double *y, double *ans){
  int t = threadIdx.x;  // thread number of a thread inside of a particular block
  int T = blockDim.x;  // total number of threads per block
  int b = blockIdx.x;  // block number of a block inside the grid
  int B = gridDim.x;  // total number of blocks per grid
  int AC = (N/T) * (N/B);  // number of assigned cells (stride)
  int index = b*T + t;  // threads index
  
  for(int i = index; i < N*N; i+=AC){
    x[i] = 5;
    y[i] = (i==(i/N+i/N*N)?1:0);
    ans[i] = (double)0.000000000000;
  }
}

// ------------------------------------------------------------------ GPUmatmul
__global__ void GPUmatmul(int N, double *x, double *y, double *ans){
  int t = threadIdx.x;  // thread number of a thread inside of a particular block *********************************************** added by Stephen Devaney in part 2
  int T = blockDim.x;  // total number of threads per block *********************************************** added by Stephen Devaney in part 2
  int b = blockIdx.x;  // block number of a block inside the grid *********************************************** added by Stephen Devaney in part 3
  int B = gridDim.x;  // total number of blocks per grid *********************************************** added by Stephen Devaney in part 3
  int AC = (N/T) * (N/B);  // number of assigned cells (stride) *********************************************** added by Stephen Devaney in part 3
  int index = b*T + t;  // threads index
  for(int i = index; i < N*N; i+=AC){  // *********************************************** modified by Stephen Devaney in part 3
      for(int j = 0; j < N; j++){
          ans[i] += x[i/N+j] * y[i/N+j*N];
      }
  }
}

// ---------------------------------------------------------------------- check
bool check(int N, double *ans){
  for(int i = 0; i < N; i++) {
    for(int j = 0; j < N; j++) {
      if(ans[i*N+j] != 20.0) return false;
    }
  }
  return true;
}

// ----------------------------------------------------------------------- MAIN
int main(void){
  // size of matrix
  int N = 1<<9; // binary left-shift: 1 * 2^9 = 512
  printf("Size of matrix (N) is %d by %d.\n", N, N);
  int iter = 3;
  clock_t t;
  int blockSize = 128;  // number of threads per block *********************************************** added by Stephen Devaney in part 3
  int numBlocks = (N+blockSize-1) / blockSize;  // number of blocks *********************************************** added by Stephen Devaney in part 3
  
  // Martices
  double *x, *y, *ans;

  // Allocate Unified Memory - accessible from both CPU and GPU *********************************************** Added by Stephen Devaney in part 1
  hipMallocManaged(&x, N * N * sizeof(double));
  hipMallocManaged(&y, N * N * sizeof(double));
  hipMallocManaged(&ans, N * N * sizeof(double));
  
  // ..........................................................................
  // Prefetch the data to the GPU  // *********************************************** added by Stephen Devaney in part 4
  int device = -1;
  hipGetDevice(&device);
  hipMemPrefetchAsync(x, N * N * sizeof(double), device, NULL);
  hipMemPrefetchAsync(y, N * N * sizeof(double), device, NULL);
  hipMemPrefetchAsync(ans, N * N * sizeof(double), device, NULL);
  
  // ..........................................................................
  // initialize x,y and ans arrays on the host
  init<<<numBlocks,blockSize>>>(N, x, y, ans);  // *********************************************** modified by Stephen Devaney in part 4

  // ..........................................................................
  double avg=0;
  std::cout<<"Starting unoptimized GPU computation"<<std::endl;
  // Run kernel on GPU
  for(int i = 0; i <= iter; i++) {
    t = clock();
    GPUmatmul<<<numBlocks,blockSize>>>(N, x, y,ans); // *********************************************** updated by Stephen Devaney in part 2 and 3
    hipDeviceSynchronize();
    t = clock() - t;
    if(i) avg += t; //we will ignore the first run
    // printf ("It took GPU-%d %f ms.\n",i,(((double)t)/CLOCKS_PER_SEC)*1000);
  }

  avg /= iter;
  avg /= CLOCKS_PER_SEC;
  avg *= 1000;
  printf("It took %lf ms on avg.\n", avg);
  if(check(N,ans)) std::cout<<"RUN OK."<<std::endl;
  else std::cout<<"RUN NOT OK."<<std::endl;

  // ..........................................................................
  
  // Free memory *********************************************** Added by Stephen Devaney in part 1
  hipFree(x);
  hipFree(y);
  hipFree(ans);

  return 0;
}
/* EOF */
