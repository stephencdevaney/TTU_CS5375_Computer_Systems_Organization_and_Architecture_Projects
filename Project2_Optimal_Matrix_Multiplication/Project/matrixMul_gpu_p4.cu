#include "hip/hip_runtime.h"
/*
 * _MATRIXMUL_GPU_CU_
 *
 * Orginal code by Mert SIDE
 * Updates by Stephen Devaney
 * 2022
 *
 * CS5375 Computer Systems Organization and Architecture 
 * Guest Lecture: GPU Programming
 *
 * Multiplying two matrices on the GPU
 *
 */

#include <iostream>
#include <stdio.h>
#include <stdlib.h>

// ------------------------------------------------------------------ GPUmatmul (unified memory) // *********************************************** added by Stephen Devaney in part 4
__global__
void init(int N, double *x, double *y, double *ans){
  for (int i = 0; i < N; i++) {
    for(int j = 0; j < N; j++) {
      x[i*N+j] = 5;
      y[i*N+j] = (i==j?1:0);
      ans[i*N+j] = (double)0.000000000000;
    }
  }
}


// ------------------------------------------------------------------ GPUmatmul
__global__
void GPUmatmul(int N, double *x, double *y, double *ans){
  int t = threadIdx.x;  // thread number of a thread inside of a particular block *********************************************** added by Stephen Devaney in part 2
  int T = blockDim.x;  // total number of threads per block *********************************************** added by Stephen Devaney in part 2
  int b = blockIdx.x;  // block number of a block inside the grid *********************************************** added by Stephen Devaney in part 3
  int B = gridDim.x;  // total number of blocks per grid *********************************************** added by Stephen Devaney in part 3

  for(int i = b; i < N; i+=B){  // *********************************************** modified by Stephen Devaney in part 2 and 3
    for(int j = t; j < N; j+=T){
      for(int k = 0; k < N; k++){
        ans[i*N+j] += x[i*N+k] * y[k*N+j];
      }
    }
  }
}


// ---------------------------------------------------------------------- check
bool check(int N, double *ans){
  for(int i = 0; i < N; i++) {
    for(int j = 0; j < N; j++) {
      if(ans[i*N+j] != 20.0) return false;
    }
  }
  return true;
}


// ----------------------------------------------------------------------- MAIN
int main(void){
  // size of matrix
  int N = 1<<9; // binary left-shift: 1 * 2^9 = 512
  printf("Size of matrix (N) is %d by %d.\n", N, N);
  int iter = 3;
  clock_t t;
  int blockSize = 128;  // number of threads per block *********************************************** added by Stephen Devaney in part 3
  int numBlocks = (N+blockSize-1) / blockSize;  // number of blocks *********************************************** added by Stephen Devaney in part 3
  
  // Martices
  double *x, *y, *ans;

  // Allocate Unified Memory - accessible from both CPU and GPU *********************************************** Added by Stephen Devaney in part 1
  hipMallocManaged(&x, N * N * sizeof(double));
  hipMallocManaged(&y, N * N * sizeof(double));
  hipMallocManaged(&ans, N * N * sizeof(double));
  
  // ..........................................................................
  // Prefetch the data to the GPU
  int device = -1;
  hipGetDevice(&device);
  hipMemPrefetchAsync(x, N * N * sizeof(double), device, NULL);
  hipMemPrefetchAsync(y, N * N * sizeof(double), device, NULL);
  hipMemPrefetchAsync(ans, N * N * sizeof(double), device, NULL);
  
  // ..........................................................................
  // initialize x,y and ans arrays on the host
  init<<<numBlocks,blockSize>>>(N, x, y, ans);
  
  // ..........................................................................
  double avg=0;
  std::cout<<"Starting optimized GPU computation"<<std::endl;
  // Run kernel on GPU
  for(int i = 0; i <= iter; i++) {
    t = clock();
    GPUmatmul<<<numBlocks,blockSize>>>(N, x, y,ans); // *********************************************** updated by Stephen Devaney in part 2 and 3
    hipDeviceSynchronize();
    t = clock() - t;
    if(i) avg += t; //we will ignore the first run
    // printf ("It took GPU-%d %f ms.\n",i,(((double)t)/CLOCKS_PER_SEC)*1000);
  }

  avg /= iter;
  avg /= CLOCKS_PER_SEC;
  avg *= 1000;
  printf("It took %lf ms on avg.\n", avg);
  if(check(N,ans)) std::cout<<"RUN OK."<<std::endl;
  else std::cout<<"RUN NOT OK."<<std::endl;

  // ..........................................................................
  
  // Free memory *********************************************** Added by Stephen Devaney in part 1
  hipFree(x);
  hipFree(y);
  hipFree(ans);

  return 0;
}
/* EOF */
