/*
 * _MATRIXMUL_GPU_CU_
 *
 * Orginal code by Mert SIDE
 * Updates by Stephen Devaney
 * 2022
 *
 * CS5375 Computer Systems Organization and Architecture 
 * Guest Lecture: GPU Programming
 *
 * Multiplying two matrices on the GPU
 *
 */


#include <hip/hip_runtime.h>
#include <iostream>
#include <stdio.h>
#include <stdlib.h>

// ------------------------------------------------------------------ GPUmatmul
__global__
void GPUmatmul(int N, double *x, double *y, double *ans){
  int t = threadIdx.x;  // thread number of a thread inside of a particular block *********************************************** added by Stephen Devaney in part 2
  int T = blockDim.x;  // total number of threads per block *********************************************** added by Stephen Devaney in part 2
  for(int i = t; i < N * N; i+=T){
      for(int j = 0; j < N; j++){
          ans[i] += x[i/N+j] * y[i/N+j*N];
      }
  }
//  for(int i = t; i < N * N; i+=T){
//    for(int j = 0; j < N; j++) {
//      for(int k = 0; k < N; k++) {
//        ans[i*N+j] += (x[i*N+k] * y[k*N+j]);
//      }
//    }
//  }
}

// ---------------------------------------------------------------------- check
bool check(int N, double *ans){
  for(int i = 0; i < N; i++) {
    for(int j = 0; j < N; j++) {
      if(ans[i*N+j] != 20.0) return false;
    }
  }
  return true;
}

// ----------------------------------------------------------------------- MAIN
int main(void){
  // size of matrix
  int N = 1<<9; // binary left-shift: 1 * 2^9 = 512
  printf("Size of matrix (N) is %d by %d.\n", N, N);
  int iter = 3;
  clock_t t;
  
  // Martices
  double *x, *y, *ans;

  // Allocate Unified Memory - accessible from both CPU and GPU *********************************************** Added by Stephen Devaney in part 1
  hipMallocManaged(&x, N * N * sizeof(double));
  hipMallocManaged(&y, N * N * sizeof(double));
  hipMallocManaged(&ans, N * N * sizeof(double));

  // ..........................................................................
  // initialize x,y and ans arrays on the host
  for (int i = 0; i < N; i++) {
    for(int j = 0; j < N; j++) {
      x[i*N+j] = 5;
      y[i*N+j] = (i==j?1:0);
      ans[i*N+j] = (double)0.000000000000;
    }
  }

  // ..........................................................................
  double avg=0;
  std::cout<<"Starting unoptimized GPU computation"<<std::endl;
  // Run kernel on GPU
  for(int i = 0; i <= iter; i++) {
    t = clock();
    GPUmatmul<<<1,256>>>(N, x, y,ans); // *********************************************** updated by Stephen Devaney in part 2
    hipDeviceSynchronize();
    t = clock() - t;
    if(i) avg += t; //we will ignore the first run
    // printf ("It took GPU-%d %f ms.\n",i,(((double)t)/CLOCKS_PER_SEC)*1000);
  }

  avg /= iter;
  avg /= CLOCKS_PER_SEC;
  avg *= 1000;
  printf("It took %lf ms on avg.\n", avg);
  if(check(N,ans)) std::cout<<"RUN OK."<<std::endl;
  else std::cout<<"RUN NOT OK."<<std::endl;

  // ..........................................................................
  
  // Free memory *********************************************** Added by Stephen Devaney in part 1
  hipFree(x);
  hipFree(y);
  hipFree(ans);

  return 0;
}
/* EOF */